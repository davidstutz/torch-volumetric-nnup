#include "hip/hip_runtime.h"
#include "luaT.h"
#include "THC.h"
#include "utils.h"

#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>


/*
 * Description:
 */

__device__ int translate_idx(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_y, int scale_factor_x)
{
  int x, y, t, z, w;
  

  w = ii % d4;
  ii = ii/d4;
  z = ii % d3;
  ii = ii/d3;
  t = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w/scale_factor_x;
  z = z/scale_factor_y;
  t = t/scale_factor_t;
  d2 /= scale_factor_t;
  d3 /= scale_factor_y;
  d4 /= scale_factor_x;
  return (((((x*d1+y)*d2)+t)*d3)+z)*d4+w;

}
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_y, int scale_factor_x, int off_time, int off_x, int off_y)
{
  /* d1 = channel
    d2 = time
    d3, d4 = height, width
  */
  int x, y, t, z, w;
  w = ii % d4;
  ii = ii/d4;
  z = ii % d3;
  ii = ii/d3;
  t = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  t = t*scale_factor_t+off_time;
  w = w*scale_factor_x+off_x;
  z = z*scale_factor_y+off_y;
  d2 *= scale_factor_t;
  d3 *= scale_factor_y;
  d4 *= scale_factor_x;
  return (((((x*d1+y)*d2)+t)*d3)+z)*d4+w;

}

__global__ void upscale(float *input, float *output, long no_elements,
                        int scale_factor_t, int scale_factor_y, int scale_factor_x, int d1, int d2, int d3, int d4)
{
  // output offset:
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  int ipidx = translate_idx(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x);
  output[ii]=input[ipidx];
}


static int cunn_VolumetricUpSamplingNearest_updateOutput(lua_State *L)
{
  /* d1 = channel
    d2 = time
    d3, d4 = height, width
  */

  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor_zero(state, output);
  int scale_factor_t = luaT_getfieldcheckint(L, 1, "scale_factor_t");
  int scale_factor_y = luaT_getfieldcheckint(L, 1, "scale_factor_y");
  int scale_factor_x = luaT_getfieldcheckint(L, 1, "scale_factor_x");
  
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));

  input = THCudaTensor_newContiguous(state, input);
  // This is for allocating output Tensor
  long no_elements = 1;
  for(int i = 0; i < input->nDimension; i++){
    no_elements *= input->size[i];
  }
  no_elements *= (scale_factor_t * scale_factor_y * scale_factor_x);

  int d1;
  int d2;
  int d3;
  int d4;

  if (input->nDimension == 4) {
    d1 = output->size[0];
    d2 = output->size[1];
    d3 = output->size[2];
    d4 = output->size[3];
  } else {
    d1 = output->size[1];
    d2 = output->size[2];
    d3 = output->size[3];
    d4 = output->size[4];
  }

  float *input_data = THCudaTensor_data(state, input);
  float *output_data = THCudaTensor_data(state, output);

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  upscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data, no_elements, scale_factor_t, scale_factor_y, scale_factor_x, d1, d2, d3, d4);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in VolumetricUpSamplingNearest.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }

  // final cut:
  THCudaTensor_free(state, input);

  return 1;
}

/*
 * Description:
 */
__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements,
                              int scale_factor_t, int scale_factor_y, int scale_factor_x, int d1, int d2, int d3, int d4)
{
  // output offset:
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  for (int i=0; i < scale_factor_t; i++){
    for(int j=0; j < scale_factor_y; j++){
      for(int k=0; k < scale_factor_x; k++){  
        int ipidx = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, i, j, k);
        gradInput_data[ii] += gradOutput_data[ipidx];
      }
    }
  }
}


static int cunn_VolumetricUpSamplingNearest_updateGradInput(lua_State *L)
{
  /* d1 = channel
    d2 = time
    d3, d4 = height, width
  */

  THCState *state = getCutorchState(L);
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput  = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  int scale_factor_t = luaT_getfieldcheckint(L, 1, "scale_factor_t");
  int scale_factor_y = luaT_getfieldcheckint(L, 1, "scale_factor_y");
  int scale_factor_x = luaT_getfieldcheckint(L, 1, "scale_factor_x");

  THAssert(THCudaTensor_checkGPU(state, 2, gradOutput, gradInput));

  THCudaTensor_zero(state, gradInput);

  float *gradInput_data = THCudaTensor_data(state, gradInput);
  float *gradOutput_data = THCudaTensor_data(state, gradOutput);

  long no_elements = 1;
  for(int i = 0; i < gradInput->nDimension; i++){
    no_elements *= gradInput->size[i];
  }

  int d1;
  int d2;
  int d3;
  int d4;

  if (gradInput->nDimension == 3) {
    d1 = gradInput->size[0];
    d2 = gradInput->size[1];
    d3 = gradInput->size[2];
    d4 = gradInput->size[3];
  } else {
    d1 = gradInput->size[1];
    d2 = gradInput->size[2];
    d3 = gradInput->size[3];
    d4 = gradInput->size[4];
  }

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil((float)no_elements / (float)(n_xblocks * nthreads));
  if (n_yblocks > 65535) {
    THError("Input size is too large!  aborting");
  }
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  // kernel:
  downscale<<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data, no_elements,
    scale_factor_t, scale_factor_y, scale_factor_x, d1, d2, d3, d4);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in VolumetricUpSamplingNearest.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }

  return 1;
}

static const struct luaL_Reg cunn_VolumetricUpSamplingNearest__ [] = {
  {"VolumetricUpSamplingNearest_updateOutput", cunn_VolumetricUpSamplingNearest_updateOutput},
  {"VolumetricUpSamplingNearest_updateGradInput", cunn_VolumetricUpSamplingNearest_updateGradInput},
  {NULL, NULL}
};

void cunn_VolumetricUpSamplingNearest_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_VolumetricUpSamplingNearest__, "nn");
  lua_pop(L,1);
}
